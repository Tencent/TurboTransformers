#include "hip/hip_runtime.h"
// Copyright (C) 2020 THL A29 Limited, a Tencent company.
// All rights reserved.
// Licensed under the BSD 3-Clause License (the "License"); you may
// not use this file except in compliance with the License. You may
// obtain a copy of the License at
// https://opensource.org/licenses/BSD-3-Clause
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" basis,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
// implied. See the License for the specific language governing
// permissions and limitations under the License.
// See the AUTHORS file for names of contributors.

#include <hip/hip_runtime.h>

#include <numeric>
#include <stdexcept>

#include "turbo_transformers/layers/kernels/gpu_embedding_kernel.h"

namespace turbo_transformers {
namespace layers {
namespace kernels {

template <bool IsAdd>
static __global__ void lookup(float* dst, const float* embedding_table,
                              const int64_t* ids, int64_t vocab_size) {
  int64_t id = ids[blockIdx.x];
  int hidden_idx = threadIdx.x;
  int hidden_size = blockDim.x;
  // TODO(jiaruifang): There should have a checker to check the range of id.
  if (id >= vocab_size) {
    asm("trap;");
  }

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ > 300
  float val = __ldg(&embedding_table[id * hidden_size + hidden_idx]);
#else
  float val = embedding_table[id * hidden_size + hidden_idx];
#endif
  if (IsAdd) {
    dst[blockIdx.x * hidden_size + hidden_idx] += val;
  } else {
    dst[blockIdx.x * hidden_size + hidden_idx] = val;
  }
}

template <bool Add>
void GPULookupKernel(float* dst, const float* embedding_table,
                     const int64_t* ids, int64_t vocab_size,
                     int64_t hidden_size, int64_t num_ids,
                     hipStream_t stream) {
  dim3 grid(num_ids);
  dim3 block(hidden_size);
  if (block.x > 1024) {
    throw std::runtime_error(
        "GPULookupKernel currently does not support a hidden_size larger than "
        "1024");
  }
  lookup<Add>
      <<<grid, block, 0, stream>>>(dst, embedding_table, ids, vocab_size);
}

template void GPULookupKernel<true>(float* dst, const float* embedding_table,
                                    const int64_t* ids, int64_t vocab_size,
                                    int64_t hidden_size, int64_t num_ids,
                                    hipStream_t stream);
template void GPULookupKernel<false>(float* dst, const float* embedding_table,
                                     const int64_t* ids, int64_t vocab_size,
                                     int64_t hidden_size, int64_t num_ids,
                                     hipStream_t stream);
}  // namespace kernels
}  // namespace layers
}  // namespace turbo_transformers
